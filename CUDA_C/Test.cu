#define T 1024 //T is for MAX number of threads in a single block
#define N 500000000 //Amount of elements in the vector

#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

__global__ void add(int *a, int *b, int *c){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < N){
        c[i] = a[i] + b[i];
    }
}

int main(){
    int *a, *b, *c;
    int *dev_a, *dev_b, *dev_c;
    
    a = (int *)malloc(N * sizeof(int));
    b = (int *)malloc(N * sizeof(int));
    c = (int *)malloc(N * sizeof(int));
    
    hipMalloc((void **) &dev_a, N * sizeof(int));
    hipMalloc((void **) &dev_b, N * sizeof(int));
    hipMalloc((void **) &dev_c, N * sizeof(int));
    
    hipMemcpy(dev_a, a, (N * sizeof(int)), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, (N * sizeof(int)), hipMemcpyHostToDevice);
    
    add<<<((int)ceil(N / T)), 1024>>>(dev_a, dev_b, dev_c);
    hipMemcpy(c, dev_c, (N * sizeof(int)), hipMemcpyDeviceToHost);
    
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    
    return 0;
}
