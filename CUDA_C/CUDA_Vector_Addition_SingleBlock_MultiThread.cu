#define N 1024

#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void add(int *a, int *b, int *c){
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main(){
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;
    
    hipMalloc((void **) &dev_a, N * sizeof(int));
    hipMalloc((void **) &dev_b, N * sizeof(int));
    hipMalloc((void **) &dev_c, N * sizeof(int));
    
    for(int j = 0; j < N; j++){
        a[j] = 2;
        b[j] = 3;
    }
    hipMemcpy(dev_a, a, (N * sizeof(int)), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, (N * sizeof(int)), hipMemcpyHostToDevice);
    
    add<<<1, 1024>>>(dev_a, dev_b, dev_c);
    hipMemcpy(c, dev_c, (N * sizeof(int)), hipMemcpyDeviceToHost);
    
    for(int j = 0; j <= 5; j++)
        printf("\n%d",c[j]);
    
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    
    return 0;
}
