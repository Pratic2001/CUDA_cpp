#include "hip/hip_runtime.h"
#define N 512
#define BLOCK_DIM 32
#include<stdio.h>
int a[N][N], b[N][N], c[N][N];
__global__ void add(int *a, int *b, int *c){
    int row = threadIdx.x + blockDim.x * blockIdx.x;
    int col = threadIdx.y + blockDim.y * blockIdx.y;
    
    c[row][col] = a[row][col] + b[row][col];
}

int main(){
    int *dev_a, *dev_b, *dev_c;
    
    hipMalloc((void **) &dev_a, N*N * sizeof(int));
    hipMalloc((void **) &dev_b, N*N * sizeof(int));
    hipMalloc((void **) &dev_c, N*N * sizeof(int));
    
    for(int j = 0; j < N; j++){
        for(int m = 0; m < N; m++){
            a[j][m] = 2;
            b[j][m] = 3;
        }
    }
    hipMemcpy2D(dev_a, a, (N * N * sizeof(int)), hipMemcpyHostToDevice);
    hipMemcpy2D(dev_b, b, (N * N * sizeof(int)), hipMemcpyHostToDevice);
    
    dim3 dimBlock(BLOCK_DIM, BLOCK_DIM);
    dim3 dimGrid((int)ceil(N / dimBlock.x), (int)ceil(N / dimBlock.y));
    add<<<dimGrid, dimBlock>>>(dev_a, dev_b, dev_c);
    hipMemcpy(c, dev_c, (N * sizeof(int)), hipMemcpyDeviceToHost);
    
    for(int j = 0; j <= 5; j++){
        for(int m = 0; m < N; m++){
            printf("\n %d",c[j][m]);
        }
    }
    
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    
    return 0;
}
